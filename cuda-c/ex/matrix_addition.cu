
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <vector>

#define ROWS 4000  // Number of rows in the matrices
#define COLS 6000  // Number of columns in the matrices

__global__ void matrixAdd(float* A, float* B, float* C, int rows, int cols) {

    // 1d
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread computes one element of the result matrix
    if (idx < rows * cols) {
        C[idx] = A[idx] + B[idx];
    }
}

void print_matrix(const float* A, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%.2f", A[i * rows + j]);
            if (j < cols - 1) printf("\t");
        }
        printf("\n");
    }
}

// Function to generate a random number between 0 and 99
float random_number() {
    return (std::rand()*1./RAND_MAX);
}

int main() {
    // Seed the random number generator with the current time
    srand(time(NULL));  // Ensure that rand() produces different sequences each run

    // Local vectors hosted in memory, each with N elements
    // using a vector to host the matrix, in a row-wise allocation
    std::vector<float> h_A(ROWS * COLS), h_B(ROWS * COLS), h_C(ROWS * COLS);
    std::generate(h_A.begin(), h_A.end(), random_number);  // Fill vector 'A' with random number
    std::generate(h_B.begin(), h_B.end(), random_number);  // Fill vector 'B' with random number

    // Size in bytes for the ROWS x COLS matrix
    int size = ROWS * COLS * sizeof(float);  

    float *d_A, *d_B, *d_C;

    // Device memory allocation
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    int N_tpb = 256;
    int N_blocks = ceil(float(ROWS)*COLS/N_tpb);

    // Launch the kernel
    matrixAdd<<<N_blocks, N_tpb>>>(d_A, d_B, d_C, ROWS, COLS);

    // Copy the result matrix C from device to host
    hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);

    // Print part of the result matrix C for verification
    printf("Matrix C\n");
    print_matrix(h_C.data(), 10, 10);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    // free(C.data());

    return 0;
}
