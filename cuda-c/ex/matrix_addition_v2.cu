
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <vector>

#define ROWS 4000  // Number of rows in the matrices
#define COLS 6000  // Number of columns in the matrices

__global__ void matrixAdd(float* A, float* B, float* C, int rows, int cols) {

    // 1d
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread computes one element of the result matrix
    if (idx < rows * cols) {
        C[idx] = A[idx] + B[idx];
    }
}

void print_matrix(const float* A, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%.2f", A[i * rows + j]);
            if (j < cols - 1) printf("\t");
        }
        printf("\n");
    }
}

int main() {
    
    // Size in bytes for the ROWS x COLS matrix
    int size = ROWS * COLS * sizeof(float);  

    // Host memory allocation
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < ROWS * COLS; i++) {
        h_A[i] = 1.0 + (float)rand()/RAND_MAX;
        h_B[i] = 2.0 + (float)rand()/RAND_MAX;
    }

    // Device memory allocation
    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    int N_tpb = 256;
    int N_blocks = ceil(float(ROWS)*COLS/N_tpb);

    // Launch the kernel
    matrixAdd<<<N_blocks, N_tpb>>>(d_A, d_B, d_C, ROWS, COLS);

    // Copy the result matrix C from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print part of the result matrix C for verification
    printf("Matrix C\n");
    print_matrix(h_C, 10, 10);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    // free(C.data());

    return 0;
}
